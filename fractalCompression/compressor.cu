#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "quadtree.h"
#include "compressor.h"

namespace fractal_compression {

	__device__ void calcCoeffsDevice2(unsigned char* pixels, unsigned char* domainPixels, int width, int offsetPixels, int offsetDomain, int blocksize, short* brightDiffValue, float* contrastCoefficient,
		float paverage, float daverage, float b, float* snapshots, int snapshotoffset)
	{
		float a = 0;
		for (int i = 0; i < blocksize; i++)
		{
			for (int j = 0; j < blocksize; j++)
			{
				a += (domainPixels[offsetDomain + i * width + j] - daverage)*(pixels[offsetPixels + i * width + j] - paverage);
			}
		}
		*contrastCoefficient = a / b;
		*brightDiffValue = (paverage - (a / b) * daverage);
		snapshots[snapshotoffset * 9] = snapshotoffset;
		snapshots[snapshotoffset * 9 + 1] = width;
		snapshots[snapshotoffset * 9 + 2] = a;
		snapshots[snapshotoffset * 9 + 3] = offsetPixels;
		snapshots[snapshotoffset * 9 + 4] = offsetDomain;
		snapshots[snapshotoffset * 9 + 5] = *brightDiffValue;
		snapshots[snapshotoffset * 9 + 6] = *contrastCoefficient;
		snapshots[snapshotoffset * 9 + 7] = daverage;
		snapshots[snapshotoffset * 9 + 8] = b;
	}

	void calcCoeffsHost2(unsigned char* pixels, unsigned char* domainPixels, int width, int offsetPixels, int offsetDomain, int blocksize, short* brightDiffValue, float* contrastCoefficient,
		float paverage, float daverage, float b)
	{
		float a = 0;
		for (int i = 0; i < blocksize; i++)
		{
			for (int j = 0; j < blocksize; j++)
			{
				a += (domainPixels[offsetDomain + i * width + j] - daverage)*(pixels[offsetPixels + i * width + j] - paverage);
			}
		}
		*contrastCoefficient = a / b;
		*brightDiffValue = (paverage - (a / b) * daverage);
	}

	__device__ float calcDiff2(unsigned char* pixels, unsigned char* domainPixels, int width, int offsetPixels, int offsetDomain, int blocksize,
		float paverage, float daverage, float b, float* snapshots, int snapshotoffset)
	{
		float difference = 0;
		short brightDiffValue = 0;
		float contrastCoefficient = 0;
		calcCoeffsDevice2(pixels, domainPixels, width, offsetPixels, offsetDomain, blocksize, &brightDiffValue, &contrastCoefficient, paverage, daverage, b, snapshots, snapshotoffset);
		for (int i = 0; i < blocksize; i++)
		{
			for (int j = 0; j < blocksize; j++)
			{
				double baseDiff = domainPixels[offsetDomain + i * width + j] * contrastCoefficient + brightDiffValue - pixels[offsetPixels + i * width + j];
				difference += baseDiff * baseDiff;
			}
		}
		return difference;
	}

	float calcDiff2Host(unsigned char* pixels, unsigned char* domainPixels, int width, int offsetPixels, int offsetDomain, int blocksize,
		float paverage, float daverage, float b)
	{
		float difference = 0;
		short brightDiffValue = 0;
		float contrastCoefficient = 0;
		calcCoeffsHost2(pixels, domainPixels, width, offsetPixels, offsetDomain, blocksize, &brightDiffValue, &contrastCoefficient, paverage, daverage, b);
		for (int i = 0; i < blocksize; i++)
		{
			for (int j = 0; j < blocksize; j++)
			{
				double baseDiff = domainPixels[offsetDomain + i * width + j] * contrastCoefficient + brightDiffValue - pixels[offsetPixels + i * width + j];
				difference += baseDiff * baseDiff;
			}
		}
		return difference;
	}

	__global__ void pickDomain(unsigned char* pixels, unsigned char* domainPixels, int n, int m, int blocksize, int pixelOffset, float* domainAverage,
		float* domainCoeffB, float paverage, float* resultArray, float* snapshots)
	{
		int affineOffset = n * m * blocksize * blocksize;
		int affineOffsetOfSnap = n * m;
		int domainOffset;
		for (int affineTransf = 0; affineTransf < 8; affineTransf++)
		{
			domainOffset = affineOffset * affineTransf + blockIdx.y * m * blocksize * blocksize + blockIdx.x * blocksize;
			resultArray[affineTransf * n * m + blockIdx.y * m + blockIdx.x] = calcDiff2(pixels, domainPixels, m * blocksize, pixelOffset, domainOffset, blocksize,
				paverage, domainAverage[blockIdx.y * m + blockIdx.x], domainCoeffB[blockIdx.y * m + blockIdx.x], snapshots, affineTransf * n * m + blockIdx.y * m + blockIdx.x);
		}
	}

	void pickDomainHost(unsigned char* pixels, unsigned char* domainPixels, int n, int m, int blocksize, int pixelOffset, float* domainAverage,
		float* domainCoeffB, float paverage, float* resultArray)
	{
		int affineOffset = n * m * blocksize * blocksize;
		int domainOffset;
		for (int affineTransf = 0; affineTransf < 8; affineTransf++)
		{
			for (int i = 0; i < n; i++)
				for (int j = 0; j < m; j++)
				{
					domainOffset = affineOffset * affineTransf + i * m * blocksize * blocksize + j * blocksize;
					resultArray[affineTransf * n * m + i * m + j] = calcDiff2Host(pixels, domainPixels, m * blocksize, pixelOffset, domainOffset, blocksize,
						paverage, domainAverage[i * m + j], domainCoeffB[i * m + j]);
				}
		}
	}

	QuadTree* fractalCompressionStep4(unsigned char* h_pixels, int sizeX, int sizeY, int startingBlockSize)
	{
		QuadTree* codes = new QuadTree(sizeY / startingBlockSize, sizeX / startingBlockSize, startingBlockSize);
		int* candidates = new int[sizeX * sizeY / (startingBlockSize * startingBlockSize)];
		unsigned char* h_domainPixels = new unsigned char[sizeX * sizeY * 8];
		unsigned char* h_domainPixels2 = new unsigned char[sizeX * sizeY * 8];
		unsigned char* d_domainPixels;
		hipMalloc(&d_domainPixels, sizeX * sizeY * 8 * sizeof(unsigned char));
		unsigned char* d_pixels;
		hipMalloc(&d_pixels, sizeX * sizeY * sizeof(unsigned char));
		hipMemcpy(d_pixels, h_pixels, sizeX * sizeY * sizeof(unsigned char), hipMemcpyHostToDevice);
		int candiateCounter = 0;
		for (int i = 0; i < sizeY; i += startingBlockSize)
		{
			for (int j = 0; j < sizeX; j += startingBlockSize)
			{
				candidates[candiateCounter] = i * sizeX + j;
				candiateCounter++;
			}
		}
		int blocksize = startingBlockSize;
		while (candiateCounter > 0)
		{
			int n = sizeY / blocksize;
			int m = sizeX / blocksize;
			int domainCount = n * m;
			float* h_domainAverage = new float[domainCount];
			float* h_rangeAverage = new float[candiateCounter];
			float* h_domainCoeffB = new float[domainCount];
			float* h_resultsArray = new float[domainCount * 8];
			float* h_snapshots = new float[domainCount * 8 * 9];
			float* d_snapshots;
			hipMalloc(&d_snapshots, domainCount * 8 * 9 * sizeof(float));
			//alocation could be more efficient. check that
			float* d_domainAverage; //= new float[domainCount];
			hipMalloc(&d_domainAverage, domainCount * sizeof(float));
			float* d_rangeAverage;// = new float[candiateCounter];
			hipMalloc(&d_rangeAverage, candiateCounter * sizeof(float));
			float* d_domainCoeffB;// = new float[domainCount];
			hipMalloc(&d_domainCoeffB, domainCount * sizeof(float));
			float* d_resultsArray;// = new float[domainCount * 8];
			hipMalloc(&d_resultsArray, 8 * domainCount * sizeof(float));
			hipMemcpy(h_resultsArray, d_resultsArray, domainCount * 8 * sizeof(float), hipMemcpyDeviceToHost);
			for (int x = 0; (x + 2) * blocksize <= sizeX; x++)
			{
				for (int y = 0; (y + 2) * blocksize <= sizeY; y++)
				{
					int offsetxl = x * blocksize;
					int offsetyl = y * blocksize;
					unsigned char** affineTransfs = new unsigned char*[8];
					affineTransfs[0] = downsize(h_pixels, offsetxl, offsetyl, blocksize * 2, sizeX);
					affineTransfs[1] = rotate90(affineTransfs[0], blocksize);
					affineTransfs[2] = rotate180(affineTransfs[0], blocksize);
					affineTransfs[3] = rotate270(affineTransfs[0], blocksize);
					affineTransfs[4] = flipHorizontal(affineTransfs[0], blocksize);
					affineTransfs[5] = flipVertical(affineTransfs[0], blocksize);
					affineTransfs[6] = flipAlongMainDiagonal(affineTransfs[0], blocksize);
					affineTransfs[7] = flipAlongSubDiagonal(affineTransfs[0], blocksize);
					for (int i = 0; i < 8; i++)
					{
						embed(h_domainPixels, affineTransfs[i], sizeX * sizeY * i + offsetyl * sizeX + offsetxl, sizeX, blocksize);
					}
					//averageandotherconsts
					int dval = 0;
					float b = 0;
					for (int i = 0; i < blocksize; i++)
					{
						for (int j = 0; j < blocksize; j++)
						{
							dval += affineTransfs[0][i * blocksize + j];
						}
					}
					float daverage = ((float)dval) / (blocksize * blocksize);
					for (int i = 0; i < blocksize; i++)
					{
						for (int j = 0; j < blocksize; j++)
						{
							b += (affineTransfs[0][i * blocksize + j] - daverage)*(affineTransfs[0][i * blocksize + j] - daverage);
						}
					}
					h_domainAverage[y * m + x] = daverage;
					h_domainCoeffB[y * m + x] = b;
					for (int i = 0; i < 8; i++)
					{
						delete[] affineTransfs[i];
					}
					delete[] affineTransfs;
				}
			}
			hipMemcpy(d_domainPixels, h_domainPixels, sizeX * sizeY * 8 * sizeof(unsigned char), hipMemcpyHostToDevice);
			hipMemcpy(d_domainAverage, h_domainAverage, domainCount * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_domainCoeffB, h_domainCoeffB, domainCount * sizeof(float), hipMemcpyHostToDevice);
			for (int i = 0; i < candiateCounter; i++)
			{
				float paverage = 0;
				for (int j = 0; j < blocksize; j++)
				{
					for (int k = 0; k < blocksize; k++)
					{
						paverage += h_pixels[candidates[i] + j * sizeX + k];
					}
				}
				paverage /= (blocksize * blocksize);
				h_rangeAverage[i] = paverage;
			}
			hipMemcpy(d_rangeAverage, h_rangeAverage, candiateCounter * sizeof(float), hipMemcpyHostToDevice);
			int newCandidateCounter = 0;
			int* newCandidates = new int[4 * sizeX * sizeY / (blocksize * blocksize)];
			for (int i = 0; i < candiateCounter; i++)
			{
				dim3 dimBlock(n, m);//dimension count is wrong. fix later
				pickDomain <<< dimBlock, 1 >>>(d_pixels, d_domainPixels, n, m, blocksize, candidates[i], d_domainAverage, d_domainCoeffB, h_rangeAverage[i], d_resultsArray, d_snapshots);
				hipMemcpy(h_resultsArray, d_resultsArray, domainCount * 8 * sizeof(float), hipMemcpyDeviceToHost);
				hipMemcpy(h_snapshots, d_snapshots, domainCount * 8 * 9 * sizeof(float), hipMemcpyDeviceToHost);
				//pickDomainHost(h_pixels, h_domainPixels, n, m, blocksize, candidates[i], h_domainAverage, h_domainCoeffB, h_rangeAverage[i], h_resultsArray);
				float mindiff = h_resultsArray[0];
				int minj = 0;
				for (int j = 0; j < domainCount * 8; j++) {
					if (mindiff > h_resultsArray[j])
					{
						mindiff = h_resultsArray[j];
						minj = j;
					}
				}
				mindiff = mindiff / (blocksize * blocksize);
				if (blocksize < 8 || mindiff < 0) {
					BlockCode* blockCode = codes->obtainNodeStart(candidates[i] / sizeX, candidates[i] % sizeX, blocksize);
					blockCode->blockSize = blocksize;
					blockCode->xoffset = candidates[i] % sizeX;
					blockCode->yoffset = candidates[i] / sizeX;
					int affinetransf = minj / domainCount;
					int nonaffoffset = minj % domainCount;
					int offsetdY = nonaffoffset / m * blocksize;
					int offsetdX = nonaffoffset % m * blocksize;
					blockCode->transformType = affinetransf;
					blockCode->ydoffset = offsetdY;
					blockCode->xdoffset = offsetdX;
					short brightDiffValue = 0;
					float contrastCoefficient = 0;
					int offsetDomain = affinetransf * domainCount * blocksize * blocksize + offsetdY * sizeX + offsetdX;
					calcCoeffsHost2(h_pixels, h_domainPixels, sizeX, candidates[i], offsetDomain, blocksize, &brightDiffValue, &contrastCoefficient,
						h_rangeAverage[i], h_domainAverage[nonaffoffset], h_domainCoeffB[nonaffoffset]);
					blockCode->brightnessDifference = brightDiffValue;
					blockCode->contrastCoefficient = contrastCoefficient;
					codes->codeCount++;
				}
				else {
					newCandidates[newCandidateCounter++] = candidates[i];
					newCandidates[newCandidateCounter++] = candidates[i] + blocksize / 2;
					newCandidates[newCandidateCounter++] = candidates[i] + sizeX * blocksize / 2;
					newCandidates[newCandidateCounter++] = candidates[i] + sizeX * blocksize / 2 + blocksize / 2;
				}
			}
			delete[] candidates;
			candidates = newCandidates;
			candiateCounter = newCandidateCounter;
			blocksize /= 2;
			delete[] h_domainAverage;
			delete[] h_domainCoeffB;
			delete[] h_rangeAverage;
			delete[] h_resultsArray;
			hipFree(d_domainAverage);
			hipFree(d_domainCoeffB);
			hipFree(d_rangeAverage);
			hipFree(d_resultsArray);
		}
		delete[] candidates;
		delete[] h_domainPixels;
		hipFree(d_domainPixels);
		hipFree(d_pixels);
		return codes;
	}
}