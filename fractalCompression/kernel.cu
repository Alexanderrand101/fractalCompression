#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

// FractalCompressorBasic.cpp : Defines the entry point for the console application.
//

#include <fstream>
#include <iostream>
#include <iomanip>
#include <Windows.h>

typedef struct HEADEROFFCOMFILE //todo. correct the size
{
	int blueDomainCount;
	int redDomainCount;
	int greenDomainCount;
	//int startingblocksize;
};

typedef struct BLOCKCODE //todo. correct the size
{
	int xoffset;
	int yoffset;
	int xdoffset;
	int ydoffset;
	byte transformType;
	int blockSize;
	float brightnessDifference;
	float contrastCoefficient;
};

typedef struct COMPRESSEDBLOCKCODE //todo. correct the size
{
	int xdoffset;
	int ydoffset;
	byte transformType;
	int blockSize;
	float brightnessDifference;
	float contrastCoefficient;
};

typedef struct QUADNODE
{
	int blocksize;
	BLOCKCODE* blockCode;
	QUADNODE** quadNodes;
};

typedef struct QUADTREE
{
	int width;
	int height;
	int startingBlockSize;
	QUADNODE** quadNodes;
};

byte* somebytes;
int valoffset = sizeof(BITMAPFILEHEADER) + sizeof(BITMAPINFOHEADER);

byte* padtoSize(byte* pixels, int oldx, int oldy, int newx, int newy) 
{
	byte* newPixels = new byte[newx * newy];
	for (int i = 0; i < oldy; i++)
	{
		for (int j = 0; j < oldx; j++) newPixels[i * newx + j] = pixels[i * oldx + j];
		for (int j = oldx; j < newx; j++) newPixels[i * newx + j] = 0;
	}
	for (int i = oldy; i < newy; i++)
		for (int j = 0; j < newx; j++)
			newPixels[i * newx + j] = 0;
	return newPixels;
}

//first draft. needs work. also need to be able to select filetype.
int LoadPixels(const char* fname, byte** reftopixels, BITMAPFILEHEADER* fheader, BITMAPINFOHEADER* iheader)
{
	std::ifstream file(fname, std::ios::binary);
	if (!file)
	{
		std::cout << "can't open file " << fname << "\n";
		return 1;
	}
	file.read((char*)fheader, sizeof(BITMAPFILEHEADER));
	file.read((char*)iheader, sizeof(BITMAPINFOHEADER));
	if (fheader->bfType != 0x4D42)
	{
		std::cout << "file " << fname << "is not a bmp file\n";
		return 2;
	}
	somebytes = new byte[fheader->bfOffBits - valoffset];
	file.read((char*)somebytes, fheader->bfOffBits - valoffset);
	*reftopixels = new byte[iheader->biSizeImage];//��� ����� �������
	file.read((char*)*reftopixels, iheader->biSizeImage);
	return 0;
}

void SavePixels(const char* fname, byte* pixels, BITMAPFILEHEADER* fheader, BITMAPINFOHEADER* iheader)
{
	std::ofstream file(fname, std::ios::binary);
	file.write((char*)fheader, sizeof(BITMAPFILEHEADER));
	file.write((char*)iheader, sizeof(BITMAPINFOHEADER));
	file.write((char*)somebytes, fheader->bfOffBits - valoffset);
	int padsize = (4 - (iheader->biWidth * 3) % 4);
	if (padsize == 4) padsize = 0;
	byte padding = 0;
	for (int i = 0; i < iheader->biHeight; i++)
	{
		file.write((char*)pixels + i * iheader->biWidth * 3, (iheader->biWidth * 3));
		for (int j = 0; j < padsize; j++)
			file.write((char*)&padding, sizeof(byte));
	}
}

byte* rotate90(byte* block, int n)
{
	byte* newblock = new byte[n * n];
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			newblock[i * n + j] = block[n * (n - j - 1) + i];
		}
	}
	return newblock;
}

byte* rotate180(byte* block, int n)
{
	byte* newblock = new byte[n * n];
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			newblock[i * n + j] = block[n * (n - i - 1) + n - j - 1];
		}
	}
	return newblock;
}

byte* rotate270(byte* block, int n)
{
	byte* newblock = new byte[n * n];
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			newblock[i * n + j] = block[j * n + n - i - 1];
		}
	}
	return newblock;
}

byte* flipHorizontal(byte* block, int n)
{
	byte* newblock = new byte[n * n];
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			newblock[i * n + j] = block[i * n + n - j - 1];
		}
	}
	return newblock;
}

byte* flipVertical(byte* block, int n)
{
	byte* newblock = new byte[n * n];
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			newblock[i * n + j] = block[(n - i - 1) * n + j];
		}
	}
	return newblock;
}

byte* flipAlongMainDiagonal(byte* block, int n)
{
	byte* newblock = new byte[n * n];;
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			newblock[i * n + j] = block[j * n + i];
		}
	}
	return newblock;
}

byte* flipAlongSubDiagonal(byte* block, int n)
{
	byte* newblock = new byte[n * n];
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			newblock[i * n + j] = block[(n - j - 1) * n + n - i - 1];
		}
	}
	return newblock;
}

byte* downsize(byte* pixels, int xoffset, int yoffset, int n, int width)
{
	int m = n / 2;
	byte* newblock = new byte[m * m];
	for (int i = 0; i < m; i++)
	{
		for (int j = 0; j < m; j++)
		{
			newblock[i * m + j] = (pixels[(yoffset + 2 * i) * width + xoffset + 2 * j] + pixels[(yoffset + 2 * i) * width + xoffset + 2 * j + 1] +
				pixels[(yoffset + 2 * i + 1) * width + xoffset + 2 * j] + pixels[(yoffset + 2 * i + 1)* width + xoffset + 2 * j + 1]) / 4;
		}
	}
	return newblock;
}


void compareAndUpdate(double* minDifference, double difference, int* ki, int k, int* li, int l, byte* affineTransform, byte caffineTransform)
{
	if (difference < *minDifference)
	{
		*minDifference = difference;
		*ki = k;
		*li = l;
		*affineTransform = caffineTransform;
	}
}

void free2Dimensions(byte** ptr, int n)
{
	for (int i = 0; i < n; i++)
	{
		delete[] ptr[i];
	}
	delete ptr;
}


void calcCoeffs(byte* block, byte*  pixels, int offsetX, int offsetY, int n, float* brightDiffValue, float* contrastCoefficient, int width)
{
	int pval = 0;
	int dval = 0;
	float a = 0;
	float b = 0;
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			pval += pixels[(offsetY + i) * width + offsetX + j];
			dval += block[i * n + j];
		}
	}
	float daverage = ((float)dval) / (n*n);
	float paverage = ((float)pval) / (n*n);
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			a += (block[i * n + j] - daverage)*(pixels[(offsetY + i) * width + offsetX + j] - paverage);
			b += (block[i * n + j] - daverage)*(block[i * n + j] - daverage);
		}
	}
	if (a - 0.001 < 0 && b - 0.001 < 0) {
		a = 1;
		b = 1;
	}
	*contrastCoefficient = a / b;
	*brightDiffValue = (paverage - (a / b) * daverage);
}

double difference(byte* block, byte*  pixels, int offsetX, int offsetY, int n, int width)
{
	double difference = 0;
	float brightDiffValue = 0;
	float contrastCoefficient = 0;
	calcCoeffs(block, pixels, offsetX, offsetY, n, &brightDiffValue, &contrastCoefficient, width);
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			difference += pow(block[i * n + j] * contrastCoefficient + brightDiffValue - pixels[(offsetY + i) * width +  offsetX + j], 2);
		}
	}
	return difference;
}




int calculateDomainSize()
{
	return 0;//todo this thing later
}

void colorChannelSeparator(byte* pixels, byte* blue, byte* green, byte* red, int width, int height)
{
	for (int i = 0; i < height; i++)
	{
		for (int j = 0; j < width; j++)
		{
			blue[i * width + j] = pixels[i * width * 3 + j * 3];
			green[i * width + j] = pixels[i * width * 3 + j * 3 + 1];
			red[i * width + j] = pixels[i * width * 3 + j * 3 + 2];
		}
	}
}

void colorChannelCombinator(byte* pixels, byte* blue, byte* green, byte* red, int width, int height)
{
	for (int i = 0; i < height; i++)
	{
		for (int j = 0; j < width; j++)
		{
			pixels[i * width * 3 + j * 3] = blue[i * width + j];
			pixels[i * width * 3 + j * 3 + 1] = green[i * width + j];
			pixels[i * width * 3 + j * 3 + 2] = red[i * width + j];
		}
	}
}

void embed(byte* pixels, byte* toEmbed, int offset, int width, int blocksize)
{
	for (int i = 0; i < blocksize; i++)
	{
		for (int j = 0; j < blocksize; j++)
		{
			pixels[offset + i * width + j] = toEmbed[i * blocksize + j];
		}
	}
}

__device__ void calcCoeffsDevice2(byte* pixels, byte* domainPixels, int width, int offsetPixels, int offsetDomain, int blocksize, float* brightDiffValue, float* contrastCoefficient,
	float paverage, float daverage, float b, float* snapshots, int snapshotoffset)
{
	float a = 0;
	for (int i = 0; i < blocksize; i++)
	{
		for (int j = 0; j < blocksize; j++)
		{
			a += (domainPixels[offsetDomain + i * width + j] - daverage)*(pixels[offsetPixels + i * width + j] - paverage);
		}
	}
	*contrastCoefficient = a / b;
	*brightDiffValue = (paverage - (a / b) * daverage);
	snapshots[snapshotoffset * 9] = snapshotoffset;
	snapshots[snapshotoffset * 9 + 1] = width;
	snapshots[snapshotoffset * 9 + 2] = a;
	snapshots[snapshotoffset * 9 + 3] = offsetPixels;
	snapshots[snapshotoffset * 9 + 4] = offsetDomain;
	snapshots[snapshotoffset * 9 + 5] = *brightDiffValue;
	snapshots[snapshotoffset * 9 + 6] = *contrastCoefficient;
	snapshots[snapshotoffset * 9 + 7] = daverage;
	snapshots[snapshotoffset * 9 + 8] = b;
}

void calcCoeffsHost2(byte* pixels, byte* domainPixels, int width, int offsetPixels, int offsetDomain, int blocksize, float* brightDiffValue, float* contrastCoefficient,
	float paverage, float daverage, float b)
{
	float a = 0;
	for (int i = 0; i < blocksize; i++)
	{
		for (int j = 0; j < blocksize; j++)
		{
			a += (domainPixels[offsetDomain + i * width + j] - daverage)*(pixels[offsetPixels + i * width + j] - paverage);
		}
	}
	*contrastCoefficient = a / b;
	*brightDiffValue = (paverage - (a / b) * daverage);
}

__device__ float calcDiff2(byte* pixels, byte* domainPixels, int width, int offsetPixels, int offsetDomain, int blocksize,
	float paverage, float daverage, float b, float* snapshots, int snapshotoffset)
{
	float difference = 0;
	float brightDiffValue = 0;
	float contrastCoefficient = 0;
	calcCoeffsDevice2(pixels, domainPixels, width, offsetPixels, offsetDomain, blocksize, &brightDiffValue, &contrastCoefficient, paverage, daverage, b, snapshots, snapshotoffset);
	/*snapshots[snapshotoffset * 9] = snapshotoffset;
	snapshots[snapshotoffset * 9 + 1] = width;
	snapshots[snapshotoffset * 9 + 2] = blocksize;
	snapshots[snapshotoffset * 9 + 3] = offsetPixels;
	snapshots[snapshotoffset * 9 + 4] = offsetDomain;
	snapshots[snapshotoffset * 9 + 5] = brightDiffValue;
	snapshots[snapshotoffset * 9 + 6] = contrastCoefficient;
	snapshots[snapshotoffset * 9 + 7] = daverage;
	snapshots[snapshotoffset * 9 + 8] = b;*/
	for (int i = 0; i < blocksize; i++)
	{
		for (int j = 0; j < blocksize; j++)
		{
			difference += pow(domainPixels[offsetDomain + i * width + j] * contrastCoefficient + brightDiffValue - pixels[offsetPixels + i * width + j], 2);
		}
	}
	return difference;
}

float calcDiff2Host(byte* pixels, byte* domainPixels, int width, int offsetPixels, int offsetDomain, int blocksize,
	float paverage, float daverage, float b)
{
	float difference = 0;
	float brightDiffValue = 0;
	float contrastCoefficient = 0;
	calcCoeffsHost2(pixels, domainPixels, width, offsetPixels, offsetDomain, blocksize, &brightDiffValue, &contrastCoefficient, paverage, daverage, b);
	for (int i = 0; i < blocksize; i++)
	{
		for (int j = 0; j < blocksize; j++)
		{
			difference += pow(domainPixels[offsetDomain + i * width + j] * contrastCoefficient + brightDiffValue - pixels[offsetPixels + i * width + j], 2);
		}
	}
	return difference;
}

__global__ void pickDomain(byte* pixels, byte* domainPixels, int n, int m, int blocksize, int pixelOffset, float* domainAverage,
	float* domainCoeffB, float paverage, float* resultArray, float* snapshots)
{
	int affineOffset = n * m * blocksize * blocksize;
	int affineOffsetOfSnap = n * m;
	int domainOffset;
	for (int affineTransf = 0; affineTransf < 8; affineTransf++)
	{
		domainOffset = affineOffset * affineTransf + blockIdx.y * m * blocksize * blocksize + blockIdx.x * blocksize;
		resultArray[affineTransf * n * m + blockIdx.y * m + blockIdx.x] = calcDiff2(pixels, domainPixels, m * blocksize, pixelOffset, domainOffset, blocksize,
			paverage, domainAverage[blockIdx.y * m + blockIdx.x], domainCoeffB[blockIdx.y * m + blockIdx.x], snapshots, affineTransf * n * m + blockIdx.y * m + blockIdx.x);
		//snapshots[(affineTransf * n * m + blockIdx.y * m + blockIdx.x) * 9] = n;
		//snapshots[(affineTransf * n * m + blockIdx.y * m + blockIdx.x) * 9 + 1] = resultArray[affineTransf * n * m + blockIdx.y * m + blockIdx.x];
		//snapshots[(affineTransf * n * m + blockIdx.y * m + blockIdx.x) * 9 + 2] = blockIdx.y;
		//snapshots[(affineTransf * n * m + blockIdx.y * m + blockIdx.x) * 9 + 3] = blockIdx.x;
		//snapshots[(affineTransf * n * m + blockIdx.y * m + blockIdx.x) * 9 + 4] = pixelOffset;
		//snapshots[(affineTransf * n * m + blockIdx.y * m + blockIdx.x) * 9 + 5] = domainOffset;
		//snapshots[(affineTransf * n * m + blockIdx.y * m + blockIdx.x) * 9 + 6] = paverage;
		//snapshots[(affineTransf * n * m + blockIdx.y * m + blockIdx.x) * 9 + 7] = domainAverage[blockIdx.y * m + blockIdx.x];
		//snapshots[(affineTransf * n * m + blockIdx.y * m + blockIdx.x) * 9 + 8] = domainCoeffB[blockIdx.y * m + blockIdx.x];
	}
}

void pickDomainHost(byte* pixels, byte* domainPixels, int n, int m, int blocksize, int pixelOffset, float* domainAverage,
	float* domainCoeffB, float paverage, float* resultArray)
{
	int affineOffset = n * m * blocksize * blocksize;
	int domainOffset;
	for (int affineTransf = 0; affineTransf < 8; affineTransf++)
	{
		for (int i = 0; i < n; i++)
			for (int j = 0; j < m; j++)
			{
				domainOffset = affineOffset * affineTransf + i * m * blocksize * blocksize + j * blocksize;
				resultArray[affineTransf * n * m + i * m + j] = calcDiff2Host(pixels, domainPixels, m * blocksize, pixelOffset, domainOffset, blocksize,
					paverage, domainAverage[i * m + j], domainCoeffB[i * m + j]);
			}
	}
}

BLOCKCODE* obtainNode(QUADNODE* node, int offseti, int offsetj, int blocksize)
{
	if (node->blocksize == blocksize) 
	{
		if (node->blockCode == nullptr) node->blockCode = new BLOCKCODE();
		return node->blockCode;
	}
	else
	{
		if (node->quadNodes == nullptr)
		{
			node->quadNodes = new QUADNODE*[4];
			for (int i = 0; i < 4; i++) node->quadNodes[i] = nullptr;
		}
		int newblocksize = node->blocksize / 2;
		int i = offseti / newblocksize;
		int j = offsetj / newblocksize;
		if (node->quadNodes[i * 2 + j] == nullptr)
		{
			node->quadNodes[i * 2 + j] = new QUADNODE();
			node->quadNodes[i * 2 + j]->blocksize = newblocksize;
			node->quadNodes[i * 2 + j]->blockCode = nullptr;
			node->quadNodes[i * 2 + j]->quadNodes = nullptr;
		}
		return obtainNode(node->quadNodes[i * 2 + j], offseti % newblocksize,
			offsetj % newblocksize, blocksize);
	}
}

BLOCKCODE* obtainNodeStart(QUADTREE* quadtree, int offsety, int offsetx, int blocksize) 
{
	int i = offsety / quadtree->startingBlockSize;
	int j = offsetx / quadtree->startingBlockSize;
	QUADNODE** nodes = quadtree->quadNodes;
	if (nodes[i * quadtree->width + j] == nullptr)
	{
		nodes[i * quadtree->width + j] = new QUADNODE();
		nodes[i * quadtree->width + j]->blocksize = quadtree->startingBlockSize;
		nodes[i * quadtree->width + j]->blockCode = nullptr;
		nodes[i * quadtree->width + j]->quadNodes = nullptr;
	}
	return obtainNode(nodes[i * quadtree->width + j], offsety % quadtree->startingBlockSize,
		offsetx % quadtree->startingBlockSize, blocksize);
}

QUADTREE* fractalCompressionStep4(byte* h_pixels, int sizeX, int sizeY, int startingBlockSize, int* codecount)
{
	*codecount = 0;
	QUADTREE* codes = new QUADTREE();
	codes->height = sizeY / startingBlockSize;
	codes->width = sizeX / startingBlockSize;
	codes->startingBlockSize = startingBlockSize;
	codes->quadNodes = new QUADNODE*[sizeX * sizeY];
	for (int i = 0; i < sizeX * sizeY; i++) codes->quadNodes[i] = nullptr;
	int* candidates = new int[sizeX * sizeY / (startingBlockSize * startingBlockSize)];
	byte* h_domainPixels = new byte[sizeX * sizeY * 8];
	byte* h_domainPixels2 = new byte[sizeX * sizeY * 8];
	byte* d_domainPixels;
	hipMalloc(&d_domainPixels, sizeX * sizeY * 8 * sizeof(byte));
	byte* d_pixels;
	hipMalloc(&d_pixels, sizeX * sizeY * sizeof(byte));
	hipMemcpy(d_pixels, h_pixels, sizeX * sizeY * sizeof(byte), hipMemcpyHostToDevice);
	int candiateCounter = 0;
	for (int i = 0; i < sizeY; i+= startingBlockSize)
	{
		for (int j = 0; j < sizeX; j+= startingBlockSize)
		{
			candidates[candiateCounter] = i * sizeX + j;
			candiateCounter++;
		}
	}
	int blocksize = startingBlockSize;
	while (candiateCounter > 0) 
	{
		int n = sizeY / blocksize;
		int m = sizeX / blocksize;
		int domainCount = n * m;
		float* h_domainAverage = new float[domainCount];
		float* h_rangeAverage = new float[candiateCounter];
		float* h_domainCoeffB = new float[domainCount];
		float* h_resultsArray = new float[domainCount * 8];
		float* h_snapshots = new float[domainCount * 8 * 9];
		float* d_snapshots;
		hipMalloc(&d_snapshots, domainCount * 8 * 9 * sizeof(float));
		//alocation could be more efficient. check that
		float* d_domainAverage; //= new float[domainCount];
		hipMalloc(&d_domainAverage, domainCount * sizeof(float));
		float* d_rangeAverage;// = new float[candiateCounter];
		hipMalloc(&d_rangeAverage, candiateCounter * sizeof(float));
		float* d_domainCoeffB;// = new float[domainCount];
		hipMalloc(&d_domainCoeffB, domainCount * sizeof(float));
		float* d_resultsArray;// = new float[domainCount * 8];
		hipMalloc(&d_resultsArray, 8 * domainCount * sizeof(float));
		hipMemcpy(h_resultsArray, d_resultsArray, domainCount * 8 * sizeof(float), hipMemcpyDeviceToHost);
		for (int x = 0; (x + 2) * blocksize <= sizeX; x++)
		{
			for (int y = 0; (y + 2) * blocksize <= sizeY; y++)
			{
				int offsetxl = x * blocksize;
				int offsetyl = y * blocksize;
				byte** affineTransfs = new byte*[8];
				affineTransfs[0] = downsize(h_pixels, offsetxl, offsetyl, blocksize * 2, sizeX);
				affineTransfs[1] = rotate90(affineTransfs[0], blocksize);
				affineTransfs[2] = rotate180(affineTransfs[0], blocksize);
				affineTransfs[3] = rotate270(affineTransfs[0], blocksize);
				affineTransfs[4] = flipHorizontal(affineTransfs[0], blocksize);
				affineTransfs[5] = flipVertical(affineTransfs[0], blocksize);
				affineTransfs[6] = flipAlongMainDiagonal(affineTransfs[0], blocksize);
				affineTransfs[7] = flipAlongSubDiagonal(affineTransfs[0], blocksize);
				for (int i = 0; i < 8; i++)
				{
					embed(h_domainPixels, affineTransfs[i], sizeX * sizeY * i + offsetyl * sizeX + offsetxl, sizeX, blocksize);
				}
				//averageandotherconsts
				int dval = 0;
				float b = 0;
				for (int i = 0; i < blocksize; i++)
				{
					for (int j = 0; j < blocksize; j++)
					{
						dval += affineTransfs[0][i * blocksize + j];
					}
				}
				float daverage = ((float)dval) / (blocksize * blocksize);
				for (int i = 0; i < blocksize; i++)
				{
					for (int j = 0; j < blocksize; j++)
					{
						b += (affineTransfs[0][i * blocksize + j] - daverage)*(affineTransfs[0][i * blocksize + j] - daverage);
					}
				}
				h_domainAverage[y * m + x] = daverage;
				h_domainCoeffB[y * m + x] = b;
				for (int i = 0; i < 8; i++)
				{
					delete[] affineTransfs[i];
				}
				delete[] affineTransfs;
			}
		}
		hipMemcpy(d_domainPixels, h_domainPixels, sizeX * sizeY * 8 * sizeof(byte), hipMemcpyHostToDevice);
		hipMemcpy(d_domainAverage, h_domainAverage, domainCount * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_domainCoeffB, h_domainCoeffB, domainCount * sizeof(float), hipMemcpyHostToDevice);
		for (int i = 0; i < candiateCounter; i++)
		{
			float paverage = 0;
			for (int j = 0; j < blocksize; j++) 
			{
				for (int k = 0; k < blocksize; k++)
				{
					paverage += h_pixels[candidates[i] + j * sizeX + k];
				}
			}
			paverage /= (blocksize * blocksize);
			h_rangeAverage[i] = paverage;
		}
		hipMemcpy(d_rangeAverage, h_rangeAverage, candiateCounter * sizeof(float), hipMemcpyHostToDevice);
		int newCandidateCounter = 0;
		int* newCandidates = new int[4 * sizeX * sizeY / (blocksize * blocksize)];
		for (int i = 0; i < candiateCounter; i++)
		{
			dim3 dimBlock(n, m);//dimension count is wrong. fix later
			pickDomain<<<dimBlock, 1>>>(d_pixels, d_domainPixels, n, m, blocksize, candidates[i], d_domainAverage, d_domainCoeffB, h_rangeAverage[i], d_resultsArray, d_snapshots);
			hipMemcpy(h_resultsArray, d_resultsArray, domainCount * 8 * sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(h_snapshots, d_snapshots, domainCount * 8 * 9 * sizeof(float), hipMemcpyDeviceToHost);
			//pickDomainHost(h_pixels, h_domainPixels, n, m, blocksize, candidates[i], h_domainAverage, h_domainCoeffB, h_rangeAverage[i], h_resultsArray);
			float mindiff = h_resultsArray[0];
			int minj = 0;
			for (int j = 0; j < domainCount * 8; j++) {
				if (mindiff > h_resultsArray[j]) 
				{
					mindiff = h_resultsArray[j];
					minj = j;
				}
			}
			mindiff = mindiff / (blocksize * blocksize);
			if (blocksize < 8 || mindiff < 50) {
				BLOCKCODE* blockCode = obtainNodeStart(codes, candidates[i] / sizeX, candidates[i] % sizeX, blocksize);
				blockCode->blockSize = blocksize;
				blockCode->xoffset = candidates[i] % sizeX;
				blockCode->yoffset = candidates[i] / sizeX;
				int affinetransf = minj / domainCount;
				int nonaffoffset = minj % domainCount;
				int offsetdY = nonaffoffset / m * blocksize;
				int offsetdX = nonaffoffset % m * blocksize;
				blockCode->transformType = affinetransf;
				blockCode->ydoffset = offsetdY;
				blockCode->xdoffset = offsetdX;
				float brightDiffValue = 0;
				float contrastCoefficient = 0;
				int offsetDomain = affinetransf * domainCount * blocksize * blocksize + offsetdY * sizeX + offsetdX;
				calcCoeffsHost2(h_pixels, h_domainPixels, sizeX, candidates[i], offsetDomain, blocksize, &brightDiffValue, &contrastCoefficient, 
					h_rangeAverage[i], h_domainAverage[nonaffoffset], h_domainCoeffB[nonaffoffset]);
				blockCode->brightnessDifference = brightDiffValue;
				blockCode->contrastCoefficient = contrastCoefficient;
				(*codecount)++;
			}
			else {
				newCandidates[newCandidateCounter++] = candidates[i];
				newCandidates[newCandidateCounter++] = candidates[i] + blocksize / 2;
				newCandidates[newCandidateCounter++] = candidates[i] + sizeX * blocksize/2;
				newCandidates[newCandidateCounter++] = candidates[i] + sizeX * blocksize / 2 + blocksize/2;
			}
		}
		delete[] candidates;
		candidates = newCandidates;
		candiateCounter = newCandidateCounter;
		blocksize /= 2;
		delete[] h_domainAverage;
		delete[] h_domainCoeffB;
		delete[] h_rangeAverage;
		delete[] h_resultsArray;
		hipFree(d_domainAverage);
		hipFree(d_domainCoeffB);
		hipFree(d_rangeAverage);
		hipFree(d_resultsArray);
	}
	delete[] candidates;
	delete[] h_domainPixels;
	hipFree(d_domainPixels);
	hipFree(d_pixels);
	return codes;
}

void fractalCompressionStep3(byte* pixels, int offsetx, int offsety, int blocksize, int* blockamount, BLOCKCODE** blockCodes, int sizex, int sizey, int qualifer)
{
	int offsetxOfMin = 0;
	int offsetyOfMin = 0;
	double minDifference = MAXINT;
	float brightnessDifference = 0;
	byte affineTransform = 0;
	float contrastCoefficient = 0;
	bool found = false;
	for (int offsetxl = 0; offsetxl + 2 * blocksize <= sizex; offsetxl += blocksize)
	{
		for (int offsetyl = 0; offsetyl + 2 * blocksize <= sizey; offsetyl += blocksize)
		{
			byte** affineTransfs = new byte*[8];
			affineTransfs[0] = downsize(pixels, offsetxl, offsetyl, blocksize * 2, sizex);
			affineTransfs[1] = rotate90(affineTransfs[0], blocksize);
			affineTransfs[2] = rotate180(affineTransfs[0], blocksize);
			affineTransfs[3] = rotate270(affineTransfs[0], blocksize);
			affineTransfs[4] = flipHorizontal(affineTransfs[0], blocksize);
			affineTransfs[5] = flipVertical(affineTransfs[0], blocksize);
			affineTransfs[6] = flipAlongMainDiagonal(affineTransfs[0], blocksize);
			affineTransfs[7] = flipAlongSubDiagonal(affineTransfs[0], blocksize);
			double cdifference;
			for (int i = 0; i < 8; i++)
			{
				cdifference = difference(affineTransfs[i], pixels, offsetx, offsety, blocksize, sizex);
				compareAndUpdate(&minDifference, cdifference, &offsetxOfMin, offsetxl, &offsetyOfMin, offsetyl, &affineTransform, i);
			}
			for (int i = 0; i < 8; i++)
			{
				delete[] affineTransfs[i];
			}
			delete[] affineTransfs;
		}
	}
	minDifference /= (blocksize * blocksize);
	if (minDifference < qualifer || blocksize <= 4)
	{
		byte* downblock = downsize(pixels, offsetxOfMin, offsetyOfMin, blocksize * 2, sizex);
		byte* trblock = nullptr;
		switch (affineTransform)
		{
		case 1:trblock = rotate90(downblock, blocksize); break;
		case 2:trblock = rotate180(downblock, blocksize); break;
		case 3:trblock = rotate270(downblock, blocksize); break;
		case 4:trblock = flipHorizontal(downblock, blocksize); break;
		case 5:trblock = flipVertical(downblock, blocksize); break;
		case 6:trblock = flipAlongMainDiagonal(downblock, blocksize); break;
		case 7:trblock = flipAlongSubDiagonal(downblock, blocksize); break;
		}
		if (affineTransform == 0)
		{
			calcCoeffs(downblock, pixels, offsetx, offsety, blocksize, &brightnessDifference, &contrastCoefficient, sizex);
			delete[] downblock;
		}
		else
		{
			calcCoeffs(trblock, pixels, offsetx, offsety, blocksize, &brightnessDifference, &contrastCoefficient, sizex);
			delete[] downblock;
			delete[] trblock;
		}
		//in the future use initial size and check for overflow, then reallocate
		//causes a brpnt error -> blockCodes = (BLOCKCODE**)realloc(blockCodes, ((*blockamount) + 1) * sizeof(BLOCKCODE*));
		blockCodes[*blockamount] = new BLOCKCODE();
		blockCodes[*blockamount]->blockSize = blocksize;
		blockCodes[*blockamount]->brightnessDifference = brightnessDifference;
		blockCodes[*blockamount]->contrastCoefficient = contrastCoefficient;
		blockCodes[*blockamount]->transformType = affineTransform;
		blockCodes[*blockamount]->xoffset = offsetx;
		blockCodes[*blockamount]->yoffset = offsety;
		blockCodes[*blockamount]->xdoffset = offsetxOfMin;
		blockCodes[*blockamount]->ydoffset = offsetyOfMin;
		(*blockamount)++;
	}
	else
	{
		fractalCompressionStep3(pixels, offsetx, offsety, blocksize / 2, blockamount, blockCodes, sizex, sizey, qualifer);
		fractalCompressionStep3(pixels, offsetx + blocksize / 2, offsety, blocksize / 2, blockamount, blockCodes, sizex, sizey, qualifer);
		fractalCompressionStep3(pixels, offsetx, offsety + blocksize / 2, blocksize / 2, blockamount, blockCodes, sizex, sizey, qualifer);
		fractalCompressionStep3(pixels, offsetx + blocksize / 2, offsety + blocksize / 2, blocksize / 2, blockamount, blockCodes, sizex, sizey, qualifer);
	}
}




void SaveCompressed(const char* fname, BITMAPFILEHEADER* fheader, BITMAPINFOHEADER* iheader, HEADEROFFCOMFILE* cheader, BLOCKCODE** blueCode, BLOCKCODE** redCode, BLOCKCODE** greenCode)
{
	std::ofstream file(fname, std::ios::binary);
	file.write((char*)fheader, sizeof(BITMAPFILEHEADER));
	file.write((char*)iheader, sizeof(BITMAPINFOHEADER));
	file.write((char*)cheader, sizeof(HEADEROFFCOMFILE));

	for (int i = 0; i < cheader->blueDomainCount; i++)
		file.write((char*)(blueCode[i]), sizeof(BLOCKCODE));

	for (int i = 0; i < cheader->greenDomainCount; i++)
		file.write((char*)(greenCode[i]), sizeof(BLOCKCODE));

	for (int i = 0; i < cheader->redDomainCount; i++)
		file.write((char*)(redCode[i]), sizeof(BLOCKCODE));
}

int powerOf2Before(int number)
{
	int twoInPower = 1;
	while (number - twoInPower >= number / 2)
		twoInPower *= 2;
	return twoInPower;
}

void insertQuadNodeIntoArray(QUADNODE* node, BLOCKCODE** blockCodes, int* blockcounter) 
{
	for (int i = 0; i < 4; i++)
	{
		if (node->quadNodes[i]->blockCode == nullptr)
		{
			insertQuadNodeIntoArray(node->quadNodes[i], blockCodes, blockcounter);
		}
		else
		{
			blockCodes[*blockcounter] = node->quadNodes[i]->blockCode;
			(*blockcounter)++;
		}
	}
}

BLOCKCODE** quadTreeToArray(QUADTREE* qtree, int blockcount) 
{
	BLOCKCODE** blockCodes = new BLOCKCODE*[blockcount];
	int blockcounter = 0;
	for (int i = 0; i < qtree->height * qtree->width; i++) 
	{
		if (qtree->quadNodes[i]->blockCode == nullptr)
		{
			insertQuadNodeIntoArray(qtree->quadNodes[i], blockCodes, &blockcounter);
		}
		else
		{
			blockCodes[blockcounter] = qtree->quadNodes[i]->blockCode;
			blockcounter++;
		}
	}
	return blockCodes;
}

int LoadCompressed(const char* fname, BITMAPFILEHEADER* fheader, BITMAPINFOHEADER* iheader, HEADEROFFCOMFILE* cheader, BLOCKCODE*** blueCode, BLOCKCODE*** redCode, BLOCKCODE*** greenCode)
{
	std::ifstream file(fname, std::ios::binary);
	if (!file)
	{
		std::cout << "can't open file " << fname << "\n";
		return 1;
	}
	file.read((char*)fheader, sizeof(BITMAPFILEHEADER));
	file.read((char*)iheader, sizeof(BITMAPINFOHEADER));
	file.read((char*)cheader, sizeof(HEADEROFFCOMFILE));
	if (fheader->bfType != 0x4D42)
	{
		std::cout << "file " << fname << "is not a bmp file\n";
		return 2;
	}
	*blueCode = new BLOCKCODE*[cheader->blueDomainCount];//��� ����� �������
	*redCode = new BLOCKCODE*[cheader->redDomainCount];//��� ����� �������
	*greenCode = new BLOCKCODE*[cheader->greenDomainCount];//��� ����� �������
	for (int i = 0; i < cheader->blueDomainCount; i++)
	{
		(*blueCode)[i] = new BLOCKCODE();
		file.read((char*)(*blueCode)[i], sizeof(BLOCKCODE));
	}
	for (int i = 0; i < cheader->greenDomainCount; i++)
	{
		(*greenCode)[i] = new BLOCKCODE();
		file.read((char*)(*greenCode)[i], sizeof(BLOCKCODE));
	}
	for (int i = 0; i < cheader->redDomainCount; i++)
	{
		(*redCode)[i] = new BLOCKCODE();
		file.read((char*)(*redCode)[i], sizeof(BLOCKCODE));
	}
	return 0;
}

void copyPixelSquare(byte* from, byte* to, int offsetxf, int offsetyf, int offsetxt, int offsetyt, int n, float brightnessCompr, int diff, int width)
{
	for (int i = 0; i < n; i++)
		for (int j = 0; j < n; j++)
		{
			to[(i + offsetyt) * width +  j + offsetxt] = from[(i + offsetyf) * n + j + offsetxf] * brightnessCompr + diff;
			if (from[(i + offsetyf) * n + j + offsetxf] * brightnessCompr + diff > 255)
				to[(i + offsetyt) * width + j + offsetxt] = 255;
			if (from[(i + offsetyf) * n + j + offsetxf] * brightnessCompr + diff < 0)
				to[(i + offsetyt) * width + j + offsetxt] = 0;
		}
}

void pixelfromfloat(byte** pixels, float** fpixels, int sizex, int sizey)
{
	for (int i = 0; i < sizey; i++)
		for (int j = 0; j < sizex; j++)
		{
			pixels[i][j] = fpixels[i][j];
		}
}

byte* fractalDecompressionStep3(BLOCKCODE** blockCodes, int sizex, int sizey, int blockCount)
{
	byte* iterPixels = new byte[sizex * sizey];
	byte* tPixels = new byte[sizex * sizey];
	for (int i = 0; i < sizex * sizey; i++)
	{
		tPixels[i] = 0;
		iterPixels[i] = 0;
	}
	//for (int i = 0; i < sizex; i++)
	//	for (int j = 0; j < sizey; j++) {
	//		tPixels[i][j] = 255;
	//	}
	for (int iteration = 0; iteration < 100; iteration++)
	{
		for (int j = 0; j < sizey; j++)
			for (int k = 0; k < sizex; k++)
				iterPixels[j * sizex + k] = tPixels[j * sizex + k];
		for (int i = 0; i < blockCount; i++)
		{		
			BLOCKCODE* cblockCode = blockCodes[i];
			byte* affineTransformed = nullptr;
			byte* downSized = downsize(iterPixels, cblockCode->xdoffset, cblockCode->ydoffset, cblockCode->blockSize * 2, sizex);
			switch (cblockCode->transformType)//refactor this stupidity
			{
			case 0:copyPixelSquare(downSized, tPixels, 0, 0, cblockCode->xoffset, cblockCode->yoffset, cblockCode->blockSize, cblockCode->contrastCoefficient, cblockCode->brightnessDifference, sizex);  break;
			case 1:affineTransformed = rotate90(downSized, cblockCode->blockSize);
				copyPixelSquare(affineTransformed, tPixels, 0, 0, cblockCode->xoffset, cblockCode->yoffset, cblockCode->blockSize, cblockCode->contrastCoefficient, cblockCode->brightnessDifference, sizex); break;
			case 2:affineTransformed = rotate180(downSized, cblockCode->blockSize);
				copyPixelSquare(affineTransformed, tPixels, 0, 0, cblockCode->xoffset, cblockCode->yoffset, cblockCode->blockSize, cblockCode->contrastCoefficient, cblockCode->brightnessDifference, sizex); break;
			case 3:affineTransformed = rotate270(downSized, cblockCode->blockSize);
				copyPixelSquare(affineTransformed, tPixels, 0, 0, cblockCode->xoffset, cblockCode->yoffset, cblockCode->blockSize, cblockCode->contrastCoefficient, cblockCode->brightnessDifference, sizex); break;
			case 4:affineTransformed = flipHorizontal(downSized, cblockCode->blockSize);
				copyPixelSquare(affineTransformed, tPixels, 0, 0, cblockCode->xoffset, cblockCode->yoffset, cblockCode->blockSize, cblockCode->contrastCoefficient, cblockCode->brightnessDifference, sizex); break;
			case 5:affineTransformed = flipVertical(downSized, cblockCode->blockSize);
				copyPixelSquare(affineTransformed, tPixels, 0, 0, cblockCode->xoffset, cblockCode->yoffset, cblockCode->blockSize, cblockCode->contrastCoefficient, cblockCode->brightnessDifference, sizex); break;
			case 6:affineTransformed = flipAlongMainDiagonal(downSized, cblockCode->blockSize);
				copyPixelSquare(affineTransformed, tPixels, 0, 0, cblockCode->xoffset, cblockCode->yoffset, cblockCode->blockSize, cblockCode->contrastCoefficient, cblockCode->brightnessDifference, sizex); break;
			case 7:affineTransformed = flipAlongSubDiagonal(downSized, cblockCode->blockSize);
				copyPixelSquare(affineTransformed, tPixels, 0, 0, cblockCode->xoffset, cblockCode->yoffset, cblockCode->blockSize, cblockCode->contrastCoefficient, cblockCode->brightnessDifference, sizex); break;
			default: std::cout << "affine error" << '\n'; break;
			}
			if (cblockCode->transformType != 0)
			{
				delete[] affineTransformed;
			}
			delete[] downSized;
		}
	}
	return tPixels;
}

void print_matr(byte* matr, int n) {
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++)
			std::cout << std::setw(3) << (int)matr[i * n + j];
		std::cout << '\n';
	}
}

int blocksum(byte* block, int n) {
	int sum = 0;
	for (int i = 0; i < n; i++)
		for (int j = 0; j < n; j++)
			sum += block[i * n +j];
	return sum;
}

void decompressQuad(BLOCKCODE** compressedCodes, int blockSize, int offsetx, int offsety, int* counter)
{
	for (int i = 0; i < 2; i++)
	{
		for (int j = 0; j < 2; j++)
		{
			if (compressedCodes[*counter]->blockSize == blockSize)
			{
				if ((compressedCodes[*counter]->yoffset  != i * blockSize + offsety) 
					|| (compressedCodes[*counter]->xoffset != j * blockSize + offsetx))
					std::cout << "\nwe fooped up\n";
				(*counter)++;
			}
			else
			{
				decompressQuad(compressedCodes, blockSize / 2, offsetx + j * blockSize,
					offsety + i * blockSize, counter);
			}
		}
	}
}

void decompressBlockCodes(BLOCKCODE** compressedCodes, int startingBlockSize, int width, int height) 
{
	//needs more here
	int counter = 0;
	for (int i = 0; i < height; i+= startingBlockSize)
	{
		for (int j = 0; j < width; j += startingBlockSize) 
		{
			if (compressedCodes[counter]->blockSize == startingBlockSize)
			{
				if ((compressedCodes[counter]->xoffset != i) || (compressedCodes[counter]->yoffset != j))
					std::cout << "\nwe fooped up\n";
				counter++;
			}
			else
			{
				decompressQuad(compressedCodes, startingBlockSize / 2, j,
					i, &counter);
			}
		}
	}
}

int main()
{
	BITMAPFILEHEADER* fheader = nullptr;
	BITMAPINFOHEADER* iheader = nullptr;
	fheader = new BITMAPFILEHEADER();
	iheader = new BITMAPINFOHEADER();
	byte* pixels = nullptr;
	byte** reftopixels = &pixels;
	LoadPixels("glss.bmp", reftopixels, fheader, iheader);
	byte *blue, *red, *green;
	blue = new byte[iheader->biHeight * iheader->biWidth];
	red = new byte[iheader->biHeight * iheader->biWidth];
	green = new byte[iheader->biHeight * iheader->biWidth];
	colorChannelSeparator(pixels, blue, green, red, iheader->biWidth, iheader->biHeight);
	colorChannelCombinator(pixels, blue, blue, blue, iheader->biWidth, iheader->biHeight);
	SavePixels("r128b11.bmp", pixels, fheader, iheader);
	std::cout << "blue channel total: " << blocksum(blue, iheader->biHeight) << '\n';
	colorChannelCombinator(pixels, green, green, green, iheader->biWidth, iheader->biHeight);
	SavePixels("r128g11.bmp", pixels, fheader, iheader);
	std::cout << "green channel total: " << blocksum(green, iheader->biHeight) << '\n';
	colorChannelCombinator(pixels, red, red, red, iheader->biWidth, iheader->biHeight);
	SavePixels("r128r11.bmp", pixels, fheader, iheader);
	std::cout << "red channel total " << blocksum(red, iheader->biHeight) << '\n';
	HEADEROFFCOMFILE* cheader = new HEADEROFFCOMFILE();
	BLOCKCODE** blueCode = new BLOCKCODE*[4096];
	BLOCKCODE** greenCode = new BLOCKCODE*[4096];
	BLOCKCODE** redCode = new BLOCKCODE*[4096];
	BLOCKCODE** blueCode2 = new BLOCKCODE*[4096];
	BLOCKCODE** greenCode2 = new BLOCKCODE*[4096];
	BLOCKCODE** redCode2 = new BLOCKCODE*[4096];
	int initialBlockSize = powerOf2Before(min(iheader->biHeight, iheader->biWidth)) / 2;
	int blueblocks = 0;
	int redblocks = 0;
	int greenblocks = 0;
	int blueblocks2 = 0;
	int redblocks2 = 0;
	int greenblocks2 = 0;
	////blue
	//fractalCompressionStep3(blue, 0, 0, initialBlockSize, &blueblocks, blueCode, iheader->biWidth, iheader->biHeight, 50);
	//fractalCompressionStep3(blue, 0, initialBlockSize, initialBlockSize, &blueblocks, blueCode, iheader->biWidth, iheader->biHeight, 50);
	//fractalCompressionStep3(blue, initialBlockSize, 0, initialBlockSize, &blueblocks, blueCode, iheader->biWidth, iheader->biHeight, 50);
	//fractalCompressionStep3(blue, initialBlockSize, initialBlockSize, initialBlockSize, &blueblocks, blueCode, iheader->biWidth, iheader->biHeight, 50);
	QUADTREE* blueTree = fractalCompressionStep4(blue, iheader->biWidth, iheader->biHeight, 16, &blueblocks);
	blueCode = quadTreeToArray(blueTree, blueblocks);
	decompressBlockCodes(blueCode, 16, iheader->biWidth, iheader->biHeight);
	//red
	//fractalCompressionStep3(red, 0, 0, initialBlockSize, &redblocks, redCode, iheader->biWidth, iheader->biHeight, 50);
	//fractalCompressionStep3(red, 0, initialBlockSize, initialBlockSize, &redblocks, redCode, iheader->biWidth, iheader->biHeight, 50);
	//fractalCompressionStep3(red, initialBlockSize, 0, initialBlockSize, &redblocks, redCode, iheader->biWidth, iheader->biHeight, 50);
	//fractalCompressionStep3(red, initialBlockSize, initialBlockSize, initialBlockSize, &redblocks, redCode, iheader->biWidth, iheader->biHeight, 50);
	QUADTREE* redTree = fractalCompressionStep4(red, iheader->biWidth, iheader->biHeight, 16, &redblocks);
	redCode = quadTreeToArray(redTree, redblocks);
	//green
	/*fractalCompressionStep3(green, 0, 0, initialBlockSize, &greenblocks, greenCode, iheader->biWidth, iheader->biHeight, 50);
	fractalCompressionStep3(green, 0, initialBlockSize, initialBlockSize, &greenblocks, greenCode, iheader->biWidth, iheader->biHeight, 50);
	fractalCompressionStep3(green, initialBlockSize, 0, initialBlockSize, &greenblocks, greenCode, iheader->biWidth, iheader->biHeight, 50);
	fractalCompressionStep3(green, initialBlockSize, initialBlockSize, initialBlockSize, &greenblocks, greenCode, iheader->biWidth, iheader->biHeight, 50);*/
	QUADTREE* greenTree = fractalCompressionStep4(green, iheader->biWidth, iheader->biHeight, 16, &greenblocks);
	greenCode = quadTreeToArray(greenTree, greenblocks);
	cheader->blueDomainCount = blueblocks;
	cheader->greenDomainCount = greenblocks;
	cheader->redDomainCount = redblocks;
	SaveCompressed("fcompressed128_1.frc", fheader, iheader, cheader, blueCode, redCode, greenCode);
	delete fheader;
	delete[] pixels;
	delete[] blue;
	delete[] red;
	delete[] green;
	delete iheader;
	free2Dimensions((byte**)blueCode, cheader->blueDomainCount);
	free2Dimensions((byte**)redCode, cheader->redDomainCount);
	free2Dimensions((byte**)greenCode, cheader->greenDomainCount);
	delete cheader;
	//endofcompression

	//startofdecompression
	fheader = new BITMAPFILEHEADER();
	iheader = new BITMAPINFOHEADER(); 
	cheader = new HEADEROFFCOMFILE(); 
	BLOCKCODE*** ptoblueCode = new BLOCKCODE**();
	BLOCKCODE*** ptoredCode = new BLOCKCODE**();
	BLOCKCODE*** ptogreenCode = new BLOCKCODE**();
	LoadCompressed("fcompressed128_1.frc", fheader, iheader, cheader, ptoblueCode, ptoredCode, ptogreenCode);
	byte* bluePixels = fractalDecompressionStep3(*ptoblueCode, iheader->biWidth, iheader->biHeight, cheader->blueDomainCount);
	byte* redPixels = fractalDecompressionStep3(*ptoredCode, iheader->biWidth, iheader->biHeight, cheader->redDomainCount);
	byte* greenPixels = fractalDecompressionStep3(*ptogreenCode, iheader->biWidth, iheader->biHeight, cheader->greenDomainCount);	
	pixels = new byte[iheader->biSizeImage];//��� ����� �������
	colorChannelCombinator(pixels, bluePixels, greenPixels, redPixels, iheader->biWidth, iheader->biHeight);
	SavePixels("r128_11.bmp", pixels, fheader, iheader);
	std::cout << "blue channel total2: " << blocksum(bluePixels, iheader->biHeight) << '\n';
	colorChannelCombinator(pixels, bluePixels, bluePixels, bluePixels, iheader->biWidth, iheader->biHeight);
	SavePixels("r128b121.bmp", pixels, fheader, iheader);
	std::cout << "green channel total2: " << blocksum(greenPixels, iheader->biHeight) << '\n';
	colorChannelCombinator(pixels, greenPixels, greenPixels, greenPixels, iheader->biWidth, iheader->biHeight);
	SavePixels("r128g121.bmp", pixels, fheader, iheader);
	std::cout << "red channel total2: " << blocksum(redPixels, iheader->biHeight) << '\n';
	colorChannelCombinator(pixels, redPixels, redPixels, redPixels, iheader->biWidth, iheader->biHeight);
	SavePixels("r128r121.bmp", pixels, fheader, iheader);
	delete[] pixels;
	delete[] bluePixels;
	delete[] greenPixels;
	delete[] redPixels;
	int x;
	std::cin >> x;
	return 0;
}


